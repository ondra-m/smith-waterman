#include "hip/hip_runtime.h"
#include <iostream>

#include "smith_waterman_params.h"

// ===========================================================================================
// DEVICE
// ===========================================================================================



// Get value
// -------------------------------------------------------------------------------------------
__device__ void get_value(long long match, long long deletion, long long insertion, long long &value, char &direction){

  value = max((long long)0, max(match, max(deletion, insertion)));

  direction = 0;

  if(value == 0){ return; }

  if     (value == match)    { direction = 1; }
  else if(value == deletion) { direction = 2; }
  else if(value == insertion){ direction = 3; }
}



// Run
// -------------------------------------------------------------------------------------------
__global__ void run(CUDA params){

  long long match, deletion, insertion, value;

  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int x = params.iteration - id;
  int y = id * params.cells_per_thread + 1;
  int end_y = y + params.cells_per_thread;

  char direction;
  bool first=true;



  while(y < end_y && y <= params.rows_count && x >= 0 && x < params.columns_count){
    match = (first == true ? params.column.before_prev : params.column.prev)[y-1] + (params.sequence_1[x] == params.sequence_2[y-1] ? params.match : params.mismatch);
    deletion = params.column.prev[y] + params.gap_penalty;
    insertion = (first == true ? params.column.prev : params.column.current)[y-1] + params.gap_penalty;

    get_value(match, deletion, insertion, value, direction);

    params.column.current[y] = value;
    params.directions[y-1] = direction;

    y++;
    first = false;
  }
}





// ===========================================================================================
// HOST
// ===========================================================================================



// Init CUDA
// -------------------------------------------------------------------------------------------
void CUDA_init(CUDA_params &params){

  params.cuda.column.size = (params.sequence_2.size+1) * sizeof(long); // first row is 0
  params.cuda.columns_count = params.sequence_1.size;
  params.cuda.rows_count = params.sequence_2.size;

  hipMalloc( (void**)&params.cuda.sequence_1,     params.sequence_1.size );
  hipMalloc( (void**)&params.cuda.sequence_2,     params.sequence_2.size );
  hipMalloc( (void**)&params.cuda.column.current, params.cuda.column.size );
  hipMalloc( (void**)&params.cuda.column.prev,    params.cuda.column.size );
  hipMalloc( (void**)&params.cuda.column.before_prev, params.cuda.column.size );
  hipMalloc( (void**)&params.cuda.directions,     params.directions_size ); 

  hipMemcpy( params.cuda.sequence_1, params.sequence_1.data, params.sequence_1.size, hipMemcpyHostToDevice );
  hipMemcpy( params.cuda.sequence_2, params.sequence_2.data, params.sequence_2.size, hipMemcpyHostToDevice );

  hipMemset( params.cuda.column.current, 0, params.cuda.column.size );
  hipMemset( params.cuda.column.before_prev, 0, params.cuda.column.size );
}



// Pre-run CUDA
// -------------------------------------------------------------------------------------------
void CUDA_delete(CUDA_params &params){
  hipFree(params.cuda.sequence_1);
  hipFree(params.cuda.sequence_2);
  hipFree(params.cuda.column.current);
  hipFree(params.cuda.column.prev);
  hipFree(params.cuda.column.before_prev);
  hipFree(params.cuda.directions);
}



// Prepare for run CUDA
// -------------------------------------------------------------------------------------------
void CUDA_run(CUDA_params &params){

  hipMemset( params.cuda.directions, 0, params.directions_size );
  hipMemcpy( params.cuda.column.before_prev, params.cuda.column.prev, params.cuda.column.size, hipMemcpyDeviceToDevice );
  hipMemcpy( params.cuda.column.prev, params.cuda.column.current, params.cuda.column.size, hipMemcpyDeviceToDevice );
  hipMemset( params.cuda.column.current, 0, params.cuda.column.size );

  run<<<params.cuda.blocks_count, params.cuda.threads_per_block>>>(params.cuda);

  hipMemcpy( params.result.directions, params.cuda.directions, params.directions_size, hipMemcpyDeviceToHost );
  hipMemcpy( params.result.column, params.cuda.column.current, params.cuda.column.size, hipMemcpyDeviceToHost );
}
