#include "hip/hip_runtime.h"
#include <iostream>

#include "smith_waterman_params.h"

// ===========================================================================================
// DEVICE
// ===========================================================================================
__device__ void get_value(long long match, long long deletion, long long insertion, long long &value, char &direction){

  value = max((long long)0, max(match, max(deletion, insertion)));

  direction = 0;

  if(value == 0){ return; }

  if     (value == match)    { direction = 1; }
  else if(value == deletion) { direction = 2; }
  else if(value == insertion){ direction = 3; }
}

__global__ void run(CUDA params){

  long long match, deletion, insertion, value;

  int x = params.iteration;
  char direction;

  for(int y=1; y<=params.row_count; y++){
    match = params.prev_column[y-1] + (params.sequence_1[x] == params.sequence_2[y-1] ? params.match : params.mismatch);
    deletion = params.prev_column[y] + params.gap_penalty;
    insertion = params.current_column[y-1] + params.gap_penalty;

    get_value(match, deletion, insertion, value, direction);

    params.current_column[y] = value;
    params.directions[y-1] = direction;
  }
}



// Init CUDA
// -------------------------------------------------------------------------------------------
void CUDA_init(CUDA_params &params){

  params.column_size = (params.sequence_2.size+1) * sizeof(long); // first row is 0

  params.cuda.row_count = params.sequence_2.size;


  hipMalloc( (void**)&params.cuda.sequence_1,     params.sequence_1.size );
  hipMalloc( (void**)&params.cuda.sequence_2,     params.sequence_2.size );
  hipMalloc( (void**)&params.cuda.current_column, params.column_size );
  hipMalloc( (void**)&params.cuda.prev_column,    params.column_size );
  hipMalloc( (void**)&params.cuda.directions,     params.directions.size ); 

  hipMemcpy( params.cuda.sequence_1, params.sequence_1.data, params.sequence_1.size, hipMemcpyHostToDevice );
  hipMemcpy( params.cuda.sequence_2, params.sequence_2.data, params.sequence_2.size, hipMemcpyHostToDevice );

  hipMemset( params.cuda.current_column, 0, params.column_size );
}



// Pre-run CUDA
// -------------------------------------------------------------------------------------------
void CUDA_delete(CUDA_params &params){
  hipFree(params.cuda.sequence_1);
  hipFree(params.cuda.sequence_2);
  hipFree(params.cuda.current_column);
  hipFree(params.cuda.prev_column);
  hipFree(params.cuda.directions);
}



// Prepare for run CUDA
// -------------------------------------------------------------------------------------------
void CUDA_run(CUDA_params &params){

  hipMemset( params.cuda.directions, 0, params.directions.size );
  hipMemcpy( params.cuda.prev_column, params.cuda.current_column, params.column_size, hipMemcpyDeviceToDevice );

  run<<<1, 1>>>(params.cuda);

  hipMemcpy( params.directions.data, params.cuda.directions, params.directions.size, hipMemcpyDeviceToHost );

  // for(int i=0; i<8; i++){
  //   std::cout << tmp[i] << std::endl;
  // }
}
